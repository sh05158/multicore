#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <omp.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

#define CUDA 0
#define OPENMP 1

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f

#define STEPS 1000000000
#define STEP 1/STEPS


struct saxpy_functor // define functor to calculate pie
{

	__host__ __device__
		double operator()(const int& x) const {
		double temp = (x + 0.5)*STEP;

		return (4.0 / (1.0 + temp * temp));
	}
};

int main()
{
	clock_t start, end;


	start = clock();
	double x, pi, sum = 0.0;
	
	thrust::counting_iterator<int> a(0); //define counting iterator 
	double result = thrust::transform_reduce(a,a+STEPS,saxpy_functor(), 0.0, thrust::plus<double>()); // reduce all result (add) with functor 

	pi = result * STEP; // calculate pi
	printf("pi=%.8lf\n", pi);

	end = clock();//measure program execution time 
 
	printf("PI Calculation: %lf sec   with step size = %d \n", (double)(end - start) / 1000.0,STEPS );

}
