#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <omp.h>
#include <hip/hip_runtime.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

#define GRID_SIZE 128
#define BLOCK_SIZE 16

struct Sphere {
	//define Sphere with hit function
	float   r, b, g;
	float   radius;
	float   x, y, z;
	__device__ float hit(float ox, float oy, float *n) {
		float dx = ox - x;
		float dy = oy - y;
		if (dx*dx + dy * dy < radius*radius) {
			float dz = sqrtf(radius*radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}
		return -INF;
	}
};

__global__ void kernel(unsigned char *c, Sphere* s)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x; //define x axis
	int y = blockIdx.y*blockDim.y + threadIdx.y; //define y axis

	int offset = x + y * DIM;
	float ox = (x - DIM / 2);
	float oy = (y - DIM / 2);

	float r = 0, g = 0, b = 0;
	float   maxz = -INF;
	for (int i = 0; i < SPHERES; i++) { //find all Spheres to print a pixel
		float   n;
		float   t = s[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}

	c[offset * 4 + 0] = (int)(r * 255);
	c[offset * 4 + 1] = (int)(g * 255);
	c[offset * 4 + 2] = (int)(b * 255);
	c[offset * 4 + 3] = 255;

}

void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp)
{
	//ppm write function to write result.ppm file
	int i, x, y;
	fprintf(fp, "P3\n");
	fprintf(fp, "%d %d\n", xdim, ydim);
	fprintf(fp, "255\n");
	for (y = 0; y < ydim; y++) {
		for (x = 0; x < xdim; x++) {
			i = x + y * xdim;
			fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
		}
		fprintf(fp, "\n");
	}
}

hipError_t cudaRun(); // cudaRun function to help run cuda function


int main()
{

	hipError_t cudaStatus = cudaRun(); // cuda Run
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaRun failed!");
		return -1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaRun failed!");
		return 1;
	}
	   
	return 0;
}

hipError_t cudaRun()
{
	FILE *fp = fopen("result.ppm", "w"); //write empty ppm file 

	Sphere* dev_s = 0;

	unsigned char *dev_bitmap;
	unsigned char *bitmap;

	bitmap = (unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM * 4); // allocate memory to host

	hipError_t cudaStatus;

	Sphere *temp_s = (Sphere *) malloc(sizeof(Sphere) * SPHERES); //define random sphere array

	for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000;
        temp_s[i].y = rnd(2000.0f) - 1000;
        temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }

	cudaStatus = hipSetDevice(0); //set cuda device
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "CudaSetDevice failed! Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	clock_t start, end;


	start = clock();

	cudaStatus = hipMalloc((void**)&dev_s, SPHERES * sizeof(Sphere)); // memory allocate for gpu(device) with spheres
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4); // memory allocate for gpu(device) with bitmap
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_s, temp_s, SPHERES * sizeof(Sphere), hipMemcpyHostToDevice); // copy variable from host to device (spheres)
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_bitmap, bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyHostToDevice);  // copy variable from host to device (bitmap) maybe empty bitmap
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 dimGrid(GRID_SIZE, GRID_SIZE, 1); // define grid 
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1); //define block

	kernel<<<dimGrid, dimBlock>>>(dev_bitmap, dev_s); // run kernel function 

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaRun launch failed : %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(bitmap, dev_bitmap,DIM*DIM * sizeof(unsigned char)*4, hipMemcpyDeviceToHost); // copy result device to host
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	end = clock(); // measure cuda program run time 

	printf("CUDA ray tracing: %lf sec\n", (double)(end - start) / 1000.0);

	ppm_write(bitmap,DIM,DIM,fp); // ppm write 
	printf("[result.ppm] was generated.\n");

    fclose(fp);


Error:
	//free memory allocate to gpu memory
	hipFree(dev_s);
	hipFree(dev_bitmap);

	return cudaStatus;
}

